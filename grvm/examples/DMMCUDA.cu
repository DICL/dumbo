#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define CHKCUDA(val)  ({                                         \
            hipError_t v = (val);                               \
  if (v != hipSuccess) {                                        \
      const char *ename;                                         \
      ename = hipGetErrorName(v);                               \
      fprintf(stderr, "CUDA error %s(%d) at %s:%d\n",            \
              ename, v, __FILE__, __LINE__);                     \
      exit(EXIT_FAILURE);                                        \
  }                                                              \
  v;})

__global__ void kmain(int N, int *A, int *B, int *C) {
    int idx;
    int row, col;
    int s, i;

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N * N) {
        return;
    }
    row = idx / N;
    col = idx % N;

    for( s = 0, i = 0; i < N;i++) {
        s += A[row * N + i] * B[ i * N + col];
    }
    C[row * N + col] = s;
}


int main(int argc, char **argv) {
  int N;
  int *A, *B, *C;
  int blocks;
  int *dA, *dB, *dC;
  int asize;
  clock_t ts;
  
  N = (argc > 1) ? atoi(argv[1]) : 512;
  asize = N * N * sizeof (int);
  A = (int *)malloc(asize);
  B = (int *)malloc(asize);
  C = (int *)malloc(asize);

  printf("=== begin %s where N  is %d === \n", argv[0], N);
  ts = clock();
  CHKCUDA(hipMalloc(&dA, asize));
  CHKCUDA(hipMalloc(&dB, asize));
  CHKCUDA(hipMalloc(&dC, asize));
  CHKCUDA(hipMemcpy(dA, A, asize, hipMemcpyHostToDevice));
  CHKCUDA(hipMemcpy(dB, B, asize, hipMemcpyHostToDevice));
  CHKCUDA(hipMemcpy(dC, C, asize, hipMemcpyHostToDevice));
  blocks = N * N / 512;
  kmain<<<blocks, 512>>>(N, dA, dB, dC);
  CHKCUDA(hipDeviceSynchronize());
  CHKCUDA(hipMemcpy(A, dA, asize, hipMemcpyDeviceToHost));
  CHKCUDA(hipMemcpy(B, dB, asize, hipMemcpyDeviceToHost));
  CHKCUDA(hipMemcpy(C, dC, asize, hipMemcpyDeviceToHost));  

  CHKCUDA(hipFree(dA));
  CHKCUDA(hipFree(dB));
  CHKCUDA(hipFree(dC));
  printf("=== finished %s in %d ms ===\n",
         argv[0],
         (int)((clock() - ts) / (CLOCKS_PER_SEC/1000)));
  free(A);
  free(B);
  free(C);

  return 0;
}
