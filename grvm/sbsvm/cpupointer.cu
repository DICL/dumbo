
#include "cpupointer.h"

#include "hip/hip_runtime_api.h"

#ifdef DEBUG
__device__ unsigned long long MinorTime;
__device__ unsigned long long MajorTime;
__device__ unsigned long long ClearTime;
__device__ unsigned long long WbTime;
__device__ unsigned long long Minor;
__device__ unsigned long long Major;
__device__ unsigned long long Clear;
__device__ unsigned long long Wb;
__device__ unsigned long long Collision;
__device__ unsigned long long Critical;
__device__ unsigned long long Lock;
#endif

namespace cpu_pointer {
  Tag *h_tag;
  int h_size_cache;
  std::uint8_t (*h_cache)[PAGE_SIZE];
  std::uint8_t *h_wb;

  __constant__ Tag *d_tag;
  __constant__ int d_size_cache;
  __constant__ std::uint8_t (*d_cache)[PAGE_SIZE];
  __constant__ std::uint8_t *d_wb;
}

void cpu_pointer::initialize()
{
  mem_cpy::initialize();

  int device;
  checkCudaErrors(
    hipGetDevice(&device)
  );

  hipDeviceProp_t prop;
  checkCudaErrors(
    hipGetDeviceProperties(&prop, device)
  );

  std::size_t size_mem = prop.totalGlobalMem;
  h_size_cache = size_mem >> (PAGE_BIT + 1);

  #ifdef DEBUG
  // h_size_cache = 1024*2;
  fprintf(stderr, "size_cache : %d (%f MB)\n",
         h_size_cache, PAGE_SIZE/1024.0*h_size_cache/1024.0);
  #endif


  checkCudaErrors(
    hipMalloc(&h_tag, sizeof(*d_tag)*h_size_cache)
  );
  checkCudaErrors(
    hipMemset(h_tag, 0, sizeof(*d_tag)*h_size_cache)
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_tag), &h_tag, sizeof(d_tag))
  );

  checkCudaErrors(
    hipMalloc(&h_cache, PAGE_SIZE*h_size_cache)
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_cache), &h_cache, sizeof(d_cache))
  );

  checkCudaErrors(
    hipMalloc(&h_wb, sizeof(*d_wb)*h_size_cache)
  );
  checkCudaErrors(
    hipMemset(h_wb, 0, sizeof(*d_wb)*h_size_cache)
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_wb), &h_wb, sizeof(d_wb))
  );

  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_size_cache), &h_size_cache, sizeof(d_size_cache))
  );

#ifdef DEBUG
  unsigned long long zero = 0;
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(MinorTime), &zero, sizeof(MinorTime))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(MajorTime), &zero, sizeof(MajorTime))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(ClearTime), &zero, sizeof(ClearTime))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(WbTime), &zero, sizeof(WbTime))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Minor), &zero, sizeof(Minor))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Major), &zero, sizeof(Major))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Clear), &zero, sizeof(Clear))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Wb), &zero, sizeof(Wb))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Collision), &zero, sizeof(Collision))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Critical), &zero, sizeof(Critical))
  );
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(Lock), &zero, sizeof(Lock))
  );
#endif

}

void cpu_pointer::clear_cache()
{
  checkCudaErrors(
    hipMemset(h_tag, 0, sizeof(*d_tag)*h_size_cache)
  );
}

void cpu_pointer::finalize()
{

#ifdef DEBUG
  unsigned long long tmp1, tmp2;
  checkCudaErrors(
    hipMemcpyFromSymbol(&tmp1,HIP_SYMBOL(MinorTime),sizeof(MinorTime),0)
  );
  checkCudaErrors(
    hipMemcpyFromSymbol(&tmp2,HIP_SYMBOL(MajorTime),sizeof(MajorTime),0)
  );
  tmp1-=tmp2;
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(MinorTime), &tmp1,sizeof(MinorTime))
  );
  PRINT_TIME(Minor)
  PRINT_TIME(Major)
  PRINT_TIME(Clear)
  PRINT_TIME(Wb)

  unsigned long long collision;
  checkCudaErrors(
    hipMemcpyFromSymbol(&collision, HIP_SYMBOL(Collision), sizeof(Collision),0)
  );
  fprintf(stderr, "Collision : %llu\n", collision);

  unsigned long long critical;
  checkCudaErrors(
    hipMemcpyFromSymbol(&critical, HIP_SYMBOL(Critical), sizeof(Critical),0)
  );
  fprintf(stderr, "Critical : %llu\n", critical);

  unsigned long long lock;
  checkCudaErrors(
    hipMemcpyFromSymbol(&lock, HIP_SYMBOL(Lock), sizeof(Lock),0)
  );
  fprintf(stderr, "lock access : %llu\n", lock);
#endif

  std::size_t zero = 0;
  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_size_cache), &zero, sizeof(d_size_cache))
  );
  h_size_cache = zero;

  void *null = nullptr;

  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_wb), &null, sizeof(d_wb))
  );
  checkCudaErrors(
    hipFree(h_wb)
  );
  h_wb = nullptr;

  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_cache), &null, sizeof(d_cache))
  );
  checkCudaErrors(
    hipFree(h_cache)
  );
  h_cache = nullptr;

  checkCudaErrors(
    hipMemcpyToSymbol(HIP_SYMBOL(d_tag), &null, sizeof(d_tag))
  );
  checkCudaErrors(
    hipFree(h_tag)
  );
  h_tag = nullptr;

  mem_cpy::finalize();
}

void cpu_pointer::run_handler()
{
  mem_cpy::run_handler();
  wb_handler();
}

void cpu_pointer::wb_handler()
{
  #ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
  #endif

  std::uint8_t *wb = new std::uint8_t[h_size_cache];
  hipMemcpy(wb, h_wb, sizeof(*d_wb)*h_size_cache, hipMemcpyDeviceToHost);
  for (std::size_t i=0; i<h_size_cache; ++i) {
    if (wb[i] != 0) {
      std::uintptr_t page_addr;
      hipMemcpy(&page_addr, &h_tag[i].address, sizeof(page_addr), hipMemcpyDeviceToHost);
      hipMemcpy((void*)page_addr, h_cache[i], PAGE_SIZE, hipMemcpyDeviceToHost);
    }
  }
  delete[] wb;

  #ifdef DEBUG
    hipEventRecord(stop);
    checkCudaErrors(hipEventSynchronize(stop));
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    fprintf(stderr, "wb_handler : %f ms\n", milliseconds);
    hipEventDestroy(stop);
    hipEventDestroy(start);
  #endif
}

void cpu_pointer::prefetch(void *ptr, long long int size)
{
  #ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
  #endif
  
  std::uintptr_t page_addr = (std::uintptr_t)ptr & PAGE_MASK;
  size += (std::uintptr_t)ptr - page_addr;

  while (size > 0) {
    int tag_index = (page_addr >> PAGE_BIT) % h_size_cache;
    
    hipMemcpy(h_cache[tag_index], (void*)page_addr, PAGE_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(&h_tag[tag_index].address, (void*)&page_addr, sizeof(page_addr), hipMemcpyHostToDevice);

    page_addr += PAGE_SIZE;
    size -= PAGE_SIZE;
  }

  #ifdef DEBUG
    hipEventRecord(stop);
    checkCudaErrors(hipEventSynchronize(stop));
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
   fprintf(stderr, "prefetch : %f ms\n", milliseconds);
    hipEventDestroy(stop);
    hipEventDestroy(start);
  #endif
}