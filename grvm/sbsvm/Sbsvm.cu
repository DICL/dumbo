#include "org_sbsvm_Sbsvm.h"

#include <cstdlib>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>

#include "cptr.h"
#include "example.h"


#define BUFFER_SIZE 8192


static hipDevice_t device;
static hipCtx_t context;
static std::vector<hipModule_t> module;
static std::vector<hipFunction_t> function;
static std::vector<hipStream_t> stream;

extern "C" {
/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    initialize
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_initialize
  (JNIEnv *env, jobject o)
{
  // Initialize
  hipInit(0);

  // Get number of devices supporting CUDA
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    std::cerr << "There is no device supporting CUDA." << std::endl;
    std::exit(0);
  }

  // Get handle for device 0
  hipDeviceGet(&device, 0);

  // Create context
  hipCtxCreate(&context, 0, device);
    
  cpu_pointer::initialize();
  
  hipCtxPopCurrent(&context);
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    finalize
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_finalize
  (JNIEnv *env, jobject o)
{
  cpu_pointer::finalize();
}


/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    clear
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_clear
  (JNIEnv *env, jobject o)
{
  cpu_pointer::clear_cache();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    run
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_run
  (JNIEnv *env, jobject o)
{
  cpu_pointer::run_handler();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    loadModule
 * Signature: (Ljava/nio/ByteBuffer;)J
 */
JNIEXPORT jlong JNICALL Java_org_sbsvm_Sbsvm_loadModule
  (JNIEnv *env, jobject o, jobject image)
{
  const void *pImage = env->GetDirectBufferAddress(image);
  
  hipCtxPushCurrent(context);
  hipModule_t m;
  hipJitOption options[3];
  void* values[3];
  char error_log[BUFFER_SIZE];
  options[0] = hipJitOptionErrorLogBuffer;
  values[0]  = (void*)error_log;
  options[1] = hipJitOptionErrorLogBufferSizeBytes;
  values[1]  = (void*)BUFFER_SIZE;
  options[2] = hipJitOptionTargetFromContext;
  values[2]  = 0;
  int err = hipModuleLoadDataEx(&m, pImage, 3, options, values);
  if (err != hipSuccess) {
    std::cerr << "Link error:" << std::endl << error_log << std::endl;
    std::exit(0);
  }
  hipCtxPopCurrent(&context);
  
  module.push_back(m);
  return module.size()-1;
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    getFunction
 * Signature: (JLjava/nio/ByteBuffer;)J
 */
JNIEXPORT jlong JNICALL Java_org_sbsvm_Sbsvm_getFunction
  (JNIEnv *env, jobject o, jlong moduleID, jobject name)
{
  char *pName = reinterpret_cast<char*>(env->GetDirectBufferAddress(name));
  hipFunction_t f;
  hipCtxPushCurrent(context);
  hipModuleGetFunction(&f, module[moduleID], pName);
  hipCtxPopCurrent(&context);
  function.push_back(f);
  return function.size()-1;
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    createStream
 * Signature: ()J
 */
JNIEXPORT jlong JNICALL Java_org_sbsvm_Sbsvm_createStream
  (JNIEnv *env, jobject o)
{
  hipStream_t s;
  hipCtxPushCurrent(context);
  hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
  hipCtxPopCurrent(&context);
  stream.push_back(s);
  return stream.size()-1;
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    launchKernel
 * Signature: (JJJJJJJJJ)V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_launchKernel
  (JNIEnv *env, jobject o, jlong functionID, jlong gridDimX, jlong gridDimY, jlong gridDimZ, jlong blockDimX, jlong blockDimY, jlong blockDimZ, jlong sharedMemBytes, jlong streamID)
{
  hipCtxPushCurrent(context);
  hipModuleLaunchKernel(function[functionID], gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, stream[streamID], nullptr, nullptr);
  hipCtxPopCurrent(&context);
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    test
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_test
  (JNIEnv *env, jobject o)
{
  float (*A)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  float (*B)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  float (*C)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  dim3 block(32, 8);
  dim3 grid(MAT_SIZE/block.x+(MAT_SIZE%block.x!=0), MAT_SIZE/block.y+(MAT_SIZE%block.y!=0));
  gpu_client<<<grid, block>>>(A, B, C);
}

} // extern "C"
